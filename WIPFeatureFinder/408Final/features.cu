#include "hip/hip_runtime.h"
#include "features.cuh"

// a 28*28 image has 90 000 rectangular regions with dimensions >= 4x4, times 5 possible features per region
// doing much less than that because the chances every region has all 5 features is virtually 0
#define FEATURES_PER_IMAGE (70000 * 5)
#define FEATURE_BUFFER_SIZE_PER_IMAGE (FEATURES_PER_IMAGE * sizeof(feature))
#define TOTAL_FEATURE_BUFFER_SIZE (FEATURE_BUFFER_SIZE_PER_IMAGE * concImages)
#define TOTAL_FEATURE_BUFFER_COUNT (FEATURES_PER_IMAGE * concImages)
// fraction of free VRAM to use
#define FREE_VRAM_USAGE .6
// the average pixel difference to trigger a feature
#define THRESHOLD 10
#define THREADS_PER_BLOCK 128

__global__ void findFeatures(uint32_t* imageBuffer, feature* featureBuffer, uint32_t* featureIndex) {
	uint32_t imgId = blockIdx.x;
	uint32_t* img = &(imageBuffer[IMAGE_SIZE * IMAGE_SIZE * imgId]);

	// build the SAT
	if (threadIdx.x < 32)
		scan2d(img);
	__syncthreads();

	// copy the SAT to shared memory
	__shared__ uint32_t SAT[IMAGE_SIZE * IMAGE_SIZE];
	memcpy(SAT, img, IMAGE_SIZE * IMAGE_SIZE * sizeof(uint32_t));

	__syncthreads();

	// find haar-like features
	haarfinder(SAT, featureBuffer, THRESHOLD, featureIndex);
}

void printFeature(feature feat) {
	switch (feat.type) {
	case HEDGE:
		printf("Horiz Edge\n");
		break;
	case VEDGE:
		printf("Vert Edge\n");
		break;
	case HLINE:
		printf("Horiz Line\n");
		break;
	case VLINE:
		printf("Vert Line\n");
		break;
	case RECT4:
		printf("4-Rect\n");
		break;
	default:
		printf("Something else: %d\n", feat.type);
		break;
	}

	printf("Mag: %d\n", feat.mag);
	printf("(%d, %d) -> (%d, %d)\n", feat.x1, feat.y1, feat.x2, feat.y2);
}

feature* findFeatures(uint32_t* hostImageBuffer, uint32_t count, uint32_t* numFeatures) {
	// get the amount of vram we can allocate for this step
	size_t freeMem, totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
	printf("CUDA memory: Total: %d MB, free: %d MB\n", totalMem/1024/1024, freeMem/1024/1024);

	// compute number of images we can process at once
	int32_t concImages = freeMem * FREE_VRAM_USAGE / (FEATURE_BUFFER_SIZE_PER_IMAGE + IMAGE_SIZE * IMAGE_SIZE * sizeof(uint32_t));
	printf("Computing up to %d images at once using %lu MB of memory and %d kernels\n", concImages, concImages * (FEATURE_BUFFER_SIZE_PER_IMAGE + IMAGE_SIZE * IMAGE_SIZE * sizeof(uint32_t)) / 1024 / 1024, (int)ceil((float)count / concImages));

	printf("Readying kernel 0\n");

	//feature* hostFeatureBuffer = (feature*)malloc(FEATURE_BUFFER_SIZE * concImages);
	uint32_t hostFeatureIndex;
	feature* deviceFeatureBuffer;
	uint32_t* deviceImageBuffer, *deviceFeatureIndex;

	wbCheck(hipMalloc((void**)&deviceFeatureBuffer, TOTAL_FEATURE_BUFFER_SIZE));
	wbCheck(hipMalloc((void**)&deviceImageBuffer, IMAGE_SIZE * IMAGE_SIZE * concImages * sizeof(uint32_t)));
	wbCheck(hipMalloc((void**)&deviceFeatureIndex, sizeof(uint32_t)));

	uint32_t handledImages = 0;

	bool kernelRunning = concImages < count;
	bool willTerminate = false;

	// this will be expanded if necessary
	int32_t finishedFeatureBufferSize = concImages * FEATURES_PER_IMAGE;
	int32_t numFinishedFeatures = 0;
	feature* finishedFeatures = (feature*)malloc(finishedFeatureBufferSize * sizeof(feature));

	// the CUDA part

	// we want lots of shared memory, not so much L1
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	// clear feature buffer and copy first batch of images to device
	wbCheck(hipMemset(deviceFeatureBuffer, 0, TOTAL_FEATURE_BUFFER_SIZE));
	wbCheck(hipMemset(deviceImageBuffer, 0, IMAGE_SIZE * IMAGE_SIZE * concImages * sizeof(uint32_t)));
	wbCheck(hipMemset(deviceFeatureIndex, 0, sizeof(uint32_t)));
	wbCheck(hipMemcpy(deviceImageBuffer, &(hostImageBuffer[handledImages * IMAGE_SIZE * IMAGE_SIZE]), IMAGE_SIZE * IMAGE_SIZE * min(concImages, count - handledImages) * sizeof(uint32_t), hipMemcpyHostToDevice));

	dim3 dimGrid(min(concImages, count - handledImages), 1, 1);
	dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);

	printf("Launching kernel 0\n");

	findFeatures <<<dimGrid, dimBlock>>>(deviceImageBuffer, deviceFeatureBuffer, deviceFeatureIndex);

	handledImages += min(concImages, count - handledImages);

	uint32_t kernels = 0;

	do {
		kernels++;
		// copy feature buffer from device
		// hipMemcpy blocks until the previous kernel finishes
		// see how much we have to copy
		wbCheck(hipMemcpy(&hostFeatureIndex, deviceFeatureIndex, sizeof(uint32_t), hipMemcpyDeviceToHost));
		printf("Kernel %u output sized %u elements (%u MB, %u%% full)\n", kernels, hostFeatureIndex, hostFeatureIndex * sizeof(feature) / 1024 / 1024, (uint32_t)((float)hostFeatureIndex * 100 / TOTAL_FEATURE_BUFFER_COUNT));

		if (hostFeatureIndex > TOTAL_FEATURE_BUFFER_COUNT)
			printf("Buffer overflow by %u features, increase FEATURES_PER_IMAGE or THRESHOLD\n", hostFeatureIndex - TOTAL_FEATURE_BUFFER_COUNT);

		// then copy it, but make sure it'll fit first
		if (numFinishedFeatures + hostFeatureIndex > finishedFeatureBufferSize) {
			printf("Resizing host buffer to %u elements (%u MB)\n", finishedFeatureBufferSize + TOTAL_FEATURE_BUFFER_COUNT, (finishedFeatureBufferSize * sizeof(feature) + TOTAL_FEATURE_BUFFER_SIZE) / 1024 / 1024);
			finishedFeatures = (feature*)realloc(finishedFeatures, finishedFeatureBufferSize * sizeof(feature) + TOTAL_FEATURE_BUFFER_SIZE);
			finishedFeatureBufferSize += TOTAL_FEATURE_BUFFER_COUNT;
		}
		printf("Copying buffer to host\n");
		wbCheck(hipMemcpy(&(finishedFeatures[numFinishedFeatures]), deviceFeatureBuffer, hostFeatureIndex * sizeof(feature), hipMemcpyDeviceToHost));
		numFinishedFeatures += hostFeatureIndex;
		
		willTerminate = !kernelRunning;

		// if there are more images to analyze, start them doing so
		if (handledImages < count) {
			printf("Readying kernel %u\n", kernels + 1);
			// clear feature buffer and copy next batch of images to device
			wbCheck(hipMemset(deviceFeatureBuffer, 0, TOTAL_FEATURE_BUFFER_SIZE));
			wbCheck(hipMemset(deviceImageBuffer, 0, IMAGE_SIZE * IMAGE_SIZE * concImages * sizeof(uint32_t)));
			wbCheck(hipMemset(deviceFeatureIndex, 0, sizeof(uint32_t)));
			wbCheck(hipMemcpy(deviceImageBuffer, &(hostImageBuffer[handledImages * IMAGE_SIZE * IMAGE_SIZE]), IMAGE_SIZE * IMAGE_SIZE * min(concImages, count - handledImages) * sizeof(uint32_t), hipMemcpyHostToDevice));

			dim3 dimGrid(min(concImages, count - handledImages), 1, 1);
			dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);

			printf("Launching kernel %u\n", kernels + 1);

			findFeatures <<<dimGrid, dimBlock>>>(deviceImageBuffer, deviceFeatureBuffer, deviceFeatureIndex);

			handledImages += min(concImages, count - handledImages);
		} else
			kernelRunning = false;

	} while (!willTerminate);

	// cleanup, cleanup, everybody everywhere

	// C cleanup
	finishedFeatures = (feature*)realloc(finishedFeatures, numFinishedFeatures * sizeof(feature));

	// CUDA cleanup
	wbCheck(hipFree(deviceFeatureBuffer));
	wbCheck(hipFree(deviceImageBuffer));
	wbCheck(hipFree(deviceFeatureIndex));

	*numFeatures = numFinishedFeatures;
	return finishedFeatures;
}